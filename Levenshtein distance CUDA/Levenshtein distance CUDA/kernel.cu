#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <assert.h>
#include <math.h>
#include <time.h>
#define MIN(x,y) ((x) < (y) ? (x) : (y))

__global__ void LevKernal(char *Adata, char *Bdata, int slice, int z, int blen, int *NewH, int Increment, int Max)
{
	//int i = threadIdx.x;

	int y = blockIdx.y * blockDim.y + threadIdx.y;
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int i = ((gridDim.x * blockDim.x) * y) + x;

	if (i <= Max)
	{
		int match = 0;
		int mismatch = 1;

		int startIndex;
		if (z <= 0)
		{
			startIndex = slice;
		}
		else
		{
			startIndex = Increment * z + slice;
		}

		int j = startIndex + (i*Increment);

		int row = j / blen;
		int column = j % blen;

		if (row == 0 || column == 0)
		{
			return;
		}

		int score = (Adata[row - 1] == Bdata[column - 1]) ? match : mismatch;
		//NewH[j]=j;
		NewH[column + row* blen] = MIN(NewH[(column - 1) + (row - 1) * blen] + score, MIN(NewH[(column)+(row - 1) * blen] + 1, NewH[(column - 1) + (row)* blen] + 1));
	}
}

__global__ void init_rows(int *NewH, int blen)
{
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int i = ((gridDim.x * blockDim.x) * y) + x;


	int row = i / blen;
	int column = i % blen;

	if (row == 0 && column>0)
	{
		NewH[column + row* blen] = i;
	}
}

__global__ void init_columns(int *NewH, int blen)
{
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int i = ((gridDim.x * blockDim.x) * y) + x;


	int row = i / blen;
	int column = i % blen;

	if (column == 0 && row>0)
	{
		NewH[column + row* blen] = row;
	}
}

char GetNewChar(int num)
{
	char c = 'A';

	switch (num)
	{
	case 0: c = 'A'; break;
	case 1: c = 'C'; break;
	case 2: c = 'G'; break;
	case 3: c = 'T'; break;
	}
	return c;
}
int main(int argc, char * argv[])
{

	char *a;
	char *b;
	clock_t CPUbegin, CPUend;
	clock_t GPUbegin, GPUend;
	long double time_spentCPU, time_spentGPU;
	long int LENGHT = 0;
	/*for (int count = 1; count < 2; count++)
	{ */
	printf("Enter the lenght of string: ");
	scanf("%ld", &LENGHT);
	for (int count = 1; count < 2; count++)
	{
		a = (char *)malloc(LENGHT * sizeof(char));
		b = (char *)malloc(LENGHT * sizeof(char));
		long int t;
		for (t = 0; t < LENGHT; ++t)
		{
			int num = rand() % 4;

			a[t] = GetNewChar(num);
			num = rand() % 4;
			b[t] = GetNewChar(num);
			//printf("Random number: %d, char = %c\n",num,c);
		}
		a[t] = '\0';
		b[t] = '\0';

		//a ="Book";//argv[1];
		//b ="Back";//argv[2];

		//printf("a = %s\nb = %s\n",a,b);

		int i, j;
		int score;
		int alen = strlen(a) + 1;
		int blen = strlen(b) + 1;
		int *NewH;
		int *H;


		NewH = (int *)malloc(alen * blen * sizeof(int));
		H = (int *)malloc(alen * blen * sizeof(int));

		//------------------Initializing The Matricies-------------------

		int *dev_H = 0;
		char *dev_a;
		char *dev_b;

		NewH[0] = 0;
		H[0] = 0;

		CPUbegin = clock();										//begain time of CPU

		for (i = 1; i < blen; ++i)
		{
			// NewH[ i ]= i;

			H[i] = i;
		}

		for (j = 1; j < alen; j++)
		{
			//NewH[blen * j]= j;
			H[blen * j] = j;
		}

		//---------------------Filling The Matricies----------------------


		for (int slice = 0; slice < 2 * alen - 1; ++slice)
		{
			//printf("Slice %d: \n", slice);
			int z = slice < alen ? 0 : slice - alen + 1;
			for (int j = z; j <= slice - z; ++j)
			{
				int row = j;
				int column = (slice - j);

				if (row == 0 || column == 0)
				{
					continue;
				}
				score = (a[row - 1] == b[column - 1]) ? 0 : 1;
				H[(column)+row * blen] = MIN(H[(column - 1) + (row - 1) * blen] + score, MIN(H[(column)+(row - 1) * blen] + 1, H[(column - 1) + (row)* blen] + 1));

			}
		}



		//for (int row = 1; row < alen; row++)//i
		//{
		//	for (int column = 1; column < blen; column++)//j
		//	{
		//		score = (a[row - 1] == b[column - 1]) ? 0 : 1;

		//		H[(column)+row * blen] = MIN(H[(column - 1) + (row - 1) * blen] + score, MIN(H[(column)+(row - 1) * blen] + 1, H[(column - 1) + (row)* blen] + 1));
		//		//printf("row = %d\n",row-1);
		//	}
		//}
		CPUend = clock();										//End time of CPU
		time_spentCPU = (double)(CPUend - CPUbegin) / CLOCKS_PER_SEC;
		printf("CPU time E %d = %lf Sec\n", count, time_spentCPU);

		//printf("\n____________CPU_______________\n");

		//for(int r = 0 ; r < alen ; r++)   
		// {
		//  for(int c = 0 ; c < blen ; c++)
		//{
		//	 //printf("Type a number for <line: %d, column: %d>\t", i, j);
		//	printf("%3d ", H[r *blen +c]);// printf("\n");
		//}
		//     printf("\n");
		// }



		hipSetDevice(0);

		GPUbegin = clock();
		//Create memory allocation in GPU

		hipMalloc((void**)&dev_H, alen * blen * sizeof(int));
		hipMalloc((void**)&dev_a, LENGHT * sizeof(char));
		hipMalloc((void**)&dev_b, LENGHT * sizeof(char));
		//Copy all arrays to GPU memory

		//hipMemcpy(dev_H, NewH, alen * blen * sizeof(int), hipMemcpyHostToDevice);


		const int NumberOfThreads = 256;

		//__global__ void init_rows(int *NewE,int *NewH, int q,int r, int inf)
		init_rows << <NumberOfThreads, alen >> >(dev_H, blen);
		//__global__ void init_columns(int *NewF,int *NewH, int q,int r, int inf,int blen)
		init_columns << <NumberOfThreads, blen >> >(dev_H, blen);


		//hipMemcpy(H, dev_H,  alen * blen  * sizeof(int), hipMemcpyDeviceToHost);
		hipMemcpy(dev_a, a, LENGHT * sizeof(char), hipMemcpyHostToDevice);
		hipMemcpy(dev_b, b, LENGHT * sizeof(char), hipMemcpyHostToDevice);


		//Set GPU for parallel working

		int size = (int)ceil((float)blen / (float)NumberOfThreads);
		int Increment = alen - 1;
		//begain time of GPU
		int MemSize = alen * blen;
		for (int slice = 0; slice < 2 * alen - 1; ++slice)
		{
			int z = slice < alen ? 0 : slice - alen + 1;//CPU
			size = (int)ceil((float)((slice - 2 * z) + 1));

			LevKernal << <NumberOfThreads, MemSize >> >(dev_a, dev_b, slice, z, alen, dev_H, Increment, size);
		}

		hipMemcpy(NewH, dev_H, alen * blen * sizeof(int), hipMemcpyDeviceToHost);
		//End time of GPU
		/*hipMemcpy(a, dev_a,  LENGHT * sizeof(char), hipMemcpyDeviceToHost);
		hipMemcpy(b, dev_b, LENGHT  * sizeof(char), hipMemcpyDeviceToHost);
		printf("a = %s\nb = %s\n",a,b);*/
		hipFree(dev_a);
		hipFree(dev_b);
		hipFree(dev_H);

		GPUend = clock();

		time_spentGPU = (double)(GPUend - GPUbegin) / CLOCKS_PER_SEC;
		//(a);
		//free(b);
		//free(H);
		printf("GPU time E %d = %lf Sec\n", count, time_spentGPU);
		printf("--------------------------------------\n\n");
		//	printf("\n____________GPU_______________\n");

		//	for (int r = 0; r < alen; r++)
		//	{
		//		for (int c = 0; c < blen; c++)
		//		{
		//			//	// printf("Type a number for <line: %d, column: %d>\t", i, j);
		//			printf("%3d ", NewH[r *blen + c]);// printf("\n");
		//		}
		//		printf("\n");
		//}
		//---------------------printing the matricies---------------------


		// printf("\n____________GPU_______________\n");

		// for(int r = 0 ; r < len ; r++)   
		// {
		//   for(int c = 0 ; c < blen ; c++)
		//{
		////	// printf("Type a number for <line: %d, column: %d>\t", i, j);
		//printf("%3d ", NewH[r *blen +c]);// printf("\n");
		//}
		// printf("\n");
	}

	return (0);
}
